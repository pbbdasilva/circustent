/*
 * _CT_CUDA_CU_
 *
 * Copyright (C) 2017-2021 Tactical Computing Laboratories, LLC
 * All Rights Reserved
 * contact@tactcomplabs.com
 *
 * See LICENSE in the top level directory for licensing details
 */

#include "CT_CUDA.cuh"
#ifdef _CT_CUDA_CUH_

CT_CUDA::CT_CUDA(CTBaseImpl::CTBenchType B, CTBaseImpl::CTAtomType A) :
    CTBaseImpl("CUDA", B, A),
    Array(nullptr),
    Idx(nullptr),
    d_Array(nullptr),
    d_Idx(nullptr),
    memSize(0),
    pes(0),
    iters(0),
    elems(0),
    stride(0),
    deviceID(-1),
    deviceCount(0),
    blocksPerGrid(-1),
    threadsPerBlock(-1)
    {}

CT_CUDA::~CT_CUDA() {}

bool CT_CUDA::PrintCUDADeviceProperties(int deviceID, int deviceCount) {
    hipGetDeviceCount(&deviceCount);
    std::cout << "\nNumber of CUDA enabled devices detected: " << deviceCount << std::endl;
    if (getenv("CUDA_VISIBLE_DEVICES") == nullptr) {
        std::cout << "CUDA_VISIBLE_DEVICES environment variable not set, defaulting to hipSetDevice(1)" << std::endl;
        deviceID = hipSetDevice(1);
    }

    if (!deviceID && getenv("CUDA_VISIBLE_DEVICES") == nullptr) {
        std::cout << "No target devices detected!" << std::endl;
        return false;
    }
    return true;
}

bool CT_CUDA::ParseCUDAOpts(int argc, char **argv) {
    for (int i=1; i < argc; i++) {
        std::string s(argv[i]);

        if ( (s=="-bpg") || (s=="-blocks") || (s=="--blocks") ) {
            if ( i+1 > (argc-1) ) {
                std::cout << "Error: --blocks requires an argument" << std::endl;
                return false;
            }
            std::string P(argv[i+1]);
            blocksPerGrid = atoi(P.c_str());
            i++;
        }
        else if ((s=="-tpb") || (s=="-threads") || (s=="--threads")) {
            if ( i+1 > (argc-1) ) {
                std::cout << "Error: --threads requires an argument" << std::endl;
                return false;
            }
            std::string P(argv[i+1]);
            threadsPerBlock = atoi(P.c_str());
            i++;
        }
    }

    // sanity check the options
    if ( blocksPerGrid <= 0 ) {
        std::cout << "Error: --blocks must be greater than 0" << std::endl;
        return false;
    }
    if ( threadsPerBlock <= 0 ) {
        std::cout << "Error: --threads must be greater than 0" << std::endl;
        return false;
    }

    return true;
}

bool CT_CUDA::AllocateData(uint64_t m, uint64_t p, uint64_t i, uint64_t s) {
    // save the data
    memSize = m;
    pes = p;
    iters = i;
    stride = s;
    uint64_t idxMemSize = 2 * memSize;

    // check args
    if ( pes == 0 ) {
        std::cout << "CT_CUDA::AllocateData: `pes` cannot be 0" << std::endl;
        return false;
    }
    if ( iters == 0 ) {
        std::cout << "CT_CUDA::AllocateData `iters` cannot be 0" << std::endl;
        return false;
    }
    if ( stride == 0 ) {
        std::cout << "CT_CUDA::AllocateData `stride` cannot be 0" << std::endl;
        return false;
    }

    // calculate the number of elements
    elems = (memSize/8);
    uint64_t idxElems = (idxMemSize/8);

    // test to see whether we'll stride out of bounds
    uint64_t end = (blocksPerGrid * iters * stride) - stride;
    if ( end >= elems ) {
        std::cout << "CT_CUDA::AllocateData : `Array` is not large enough for blocksPerGrid="
        << blocksPerGrid << "; iters=" << iters << "; stride=" << stride << std::endl;
        return false;
    }

    // Allocate arrays on the host  
    Array = (uint64_t *) malloc(memSize);
    if ( Array == nullptr ) {
        std::cout << "CT_CUDA::AllocateData : 'Array' could not be allocated" << std::endl;
        free(Array);
        return false;
    }

    Idx = (uint64_t *) malloc(idxMemSize);
    if ( Idx == nullptr ) {
        std::cout << "CT_CUDA::AllocateData : 'Idx' could not be allocated" << std::endl;
        free(Array);
        free(Idx);
        return false;
    }

    // Randomize the arrays on the host
    srand(time(NULL));
    if ( this->GetBenchType() == CT_PTRCHASE ) {
        for ( unsigned i = 0; i < idxElems; i++ ) {
            Idx[i] = (uint64_t)(rand()%(idxElems - 1));
        }
    }
    else {
        for ( unsigned i = 0; i < elems; i++ ) {
            Idx[i] = (uint64_t)(rand()%(elems - 1));
        }
    }
    for ( unsigned i=0; i<elems; i++ ) {
        Array[i] = (uint64_t)(rand());
    }

    // allocate data on the target device
    if ( hipMalloc(&d_Array, memSize) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Array' could not be allocated on device" << std::endl;
        hipFree(d_Array);
        free(Array);
        free(Idx);
        return false;
    }

    if ( hipMalloc(&d_Idx, idxMemSize) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Idx' could not be alloced on device" << std::endl;
        hipFree(d_Array);
        hipFree(d_Idx);
        free(Array);
        free(Idx);
        return false;
    }


    // copy arrays from host to target device
    if ( hipMemcpy(d_Array, Array, memSize, hipMemcpyHostToDevice) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Array' could not be copied to device" << std::endl;
        hipFree(d_Array);
        hipFree(d_Idx);
        free(Array);
        free(Idx);
        return false;
    }


    if ( hipMemcpy(d_Idx, Idx, idxMemSize, hipMemcpyHostToDevice) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Idx' could not be copied to device" << std::endl;
        hipFree(d_Array);
        hipFree(d_Idx);
        free(Array);
        free(Idx);
        return false;
    }

    return true;
}

bool CT_CUDA::Execute(double &Timing, double &GAMS) {

    CTBaseImpl::CTBenchType BType   = this->GetBenchType(); // benchmark type
    CTBaseImpl::CTAtomType  AType   = this->GetAtomType();  // atomic type
    double StartTime = 0.; // start time
    double EndTime   = 0.; // end time
    double OPS       = 0.; // billions of operations

    // determine benchmark type and launch the desired kernel
    if ( BType == CT_RAND ) {
        switch ( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                RAND_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            case CT_CAS:
                StartTime = this->MySecond();
                RAND_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_STRIDE1 ) {
        switch( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                STRIDE1_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            case CT_CAS:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                STRIDE1_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_STRIDEN ) {
        switch( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                STRIDEN_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes, stride );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            case CT_CAS:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                STRIDEN_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes, stride );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_PTRCHASE ) {
        switch( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                PTRCHASE_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            case CT_CAS:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                PTRCHASE_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_SG ) {
        switch( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                SG_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(4, iters, blocksPerGrid);
                break;
            case CT_CAS:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                SG_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(4, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_CENTRAL ) {
        switch( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                CENTRAL_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            case CT_CAS:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                CENTRAL_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(1, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_SCATTER ) {
        switch( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                SCATTER_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(3, iters, blocksPerGrid);
                break;
            case CT_CAS:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                SCATTER_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(3, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_GATHER ) {
        switch( AType ) {
            case CT_ADD:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                GATHER_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(3, iters, blocksPerGrid);
                break;
            case CT_CAS:
                hipDeviceSynchronize();
                StartTime = this->MySecond();
                GATHER_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipDeviceSynchronize();
                EndTime   = this->MySecond();
                OPS = this->GAM(3, iters, blocksPerGrid);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else {
        this->ReportBenchError();
        return false;
    }
    
    Timing = this->Runtime(StartTime,EndTime);
    GAMS   = OPS/Timing;

    return true;
}

bool CT_CUDA::FreeData() {
    if ( Array ) {
        free(Array);
    }
    if ( Idx ) {
        free(Idx);
    }
    if ( d_Array ) {
        hipFree(d_Array);
    }
    if ( d_Idx ) {
        hipFree(d_Idx);
    }
    return true;
}

#endif // _CT_CUDA_CUH_

// EOF